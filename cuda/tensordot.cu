#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void launch_cublas_dot(float* A, float* B, float* C, int M, int K, int N) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // cuBLAS assumes column-major ordering by default.
    // We simulate row-major by swapping A and B and transposing the operation.
    // That means we compute: Cᵗ = Bᵗ × Aᵗ
    hipblasStatus_t status = hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // No transpose on B and A
        N, M, K,                   // Dimensions of the output matrix Cᵗ
        &alpha,
        B, N,                      // B: K×N
        A, K,                      // A: M×K
        &beta,
        C, N                       // C: M×N
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS SGEMM failed with code %d\n", status);
    }

    hipblasDestroy(handle);
}